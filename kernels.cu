#include "hip/hip_runtime.h"
#include "kernels.h"
#include "Object.h"
#include "Camera.h"

#include <stdio.h>
#include <math.h>

#include <>
#include <hiprand/hiprand_kernel.h>

__device__ float wangHash(unsigned int seed) {
	seed = (seed ^ 61) ^ (seed >> 16);
	seed *= 9;
	seed = seed ^ (seed >> 4);
	seed *= 0x27d4eb2d;
	seed = seed ^ (seed >> 15);
	return float(seed) / 4294967296.0f;
}

__device__ vec3 lighting(Material mat, vec3 lightPos, vec3 lightIntensity, vec3 point, vec3 eye, vec3 normal, SceneConfig& config) {
	vec3 colour;
	vec3 L = normalise(lightPos - point);
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	
	// ambient
	if (config.ambientLighting) {
		colour = lightIntensity * mat.colour * mat.ambient;
	}

	// diffuse
	if (config.diffuseLighting) {
		float NdotL = max(dot(normal, L), 0.0f);
		colour += (NdotL > 0.0) ? (mat.colour * lightIntensity * mat.diffuse * NdotL) : vec3();
	}

	// specular
	if (config.specularLighting) {
		if (config.areaLightSpecularEffect) {
			for (int i = 0; i < config.softShadowNum; i++) {

				unsigned int seed = (x * 1000 + y) * 1973 + i * 9277;
				float radius = config.softShadowRadius;
				float theta = 2 * 3.141592654f * wangHash(seed);
				float phi = acos(2 * wangHash(seed * 16807) - 1);
				vec3 pointOnSphere = vec3(radius * sin(phi) * cos(theta), radius * sin(phi) * sin(theta), radius * cos(phi));
				vec3 L = normalise((lightPos + pointOnSphere) - point);

				vec3 v = normalise(eye);
				vec3 r = normalise(-reflect(L, normal));
				float RdotV = max(dot(r, v), 0.0f);
				colour += mat.specular * lightIntensity * pow(RdotV, mat.shininess);

			}
		}
		else {
			vec3 v = normalise(eye);
			vec3 r = normalise(-reflect(L, normal));
			float RdotV = max(dot(r, v), 0.0f);
			colour += mat.specular * lightIntensity * pow(RdotV, mat.shininess);
		}


	}

	return colour;
}

template<typename T> __device__ bool traceRay(T* objects, int objectCount, vec3 origin, vec3 dir, RayType rayType, Hit& hit, vec3& ignore = vec3(-999, -999, -999)) {
	for (int i = 0; i < objectCount; i++) {

		if (objects[i].position != ignore) {
			float t0, t1;
			if (objects[i].hit(origin, dir, t0, t1)) {

				if (rayType == ShadowRay) {
					if (!objects[i].debug) return true;
					continue;
				}

				if (rayType == ReflectRay && objects[i].debug) continue;

				if (t0 < hit.t) {
					vec3 hitPoint = origin + t0 * dir;
					vec3 normal = objects[i].normalAt(hitPoint);
					hit = { t0, objects[i].mat, hitPoint, normal, objects[i].position, objects[i].objectType, objects[i].objectName, objects[i].debug };
				}
			}
		}
	}

	if (hit.t != 999) {
		hit.normal = normalise(hit.normal);
		return true;
	}

	return false;
}

__device__ vec3 reflectionCast2(vec3& origin, vec3& dir, Scene& scene, SceneConfig& config, vec3& ignore, int depth = 1) {
	Hit sphereHit, planeHit, boxHit, closestHit;
	bool sphereTrace = traceRay(scene.spheres, scene.sphereCount, origin, dir, ReflectRay, sphereHit);
	bool planeTrace = traceRay(scene.planes, scene.planeCount, origin, dir, ReflectRay, planeHit);
	bool boxTrace = traceRay(scene.boxes, scene.boxCount, origin, dir, ReflectRay, boxHit);
	closestHit = (sphereHit.t < planeHit.t) ? ((sphereHit.t < boxHit.t) ? sphereHit : boxHit) : ((planeHit.t < boxHit.t) ? planeHit : boxHit);


	if (sphereTrace || planeTrace || boxTrace) {
		vec3 col = lighting(closestHit.mat, scene.lights[0].position, scene.lights[0].colour, closestHit.hitPoint, scene.cam.getPosition(), normalise(closestHit.normal), config);

		if (config.renderHardShadows) {
			Hit shadowHit;
			bool shadowTrace = traceRay(scene.spheres, scene.sphereCount, closestHit.hitPoint + closestHit.normal * config.shadowBias, normalise(scene.lights[0].position - closestHit.hitPoint), ShadowRay, shadowHit);
			col = shadowTrace ? col * config.shadowIntensity : col;
		}

		return col;
	}

	return config.backgroundCol;
}

__device__ vec3 reflectionCast(vec3& origin, vec3& dir, Scene& scene, SceneConfig& config, vec3& ignore, int depth = 1) {
	Hit sphereHit, planeHit, boxHit, closestHit;
	bool sphereTrace = traceRay(scene.spheres, scene.sphereCount, origin, dir, ReflectRay, sphereHit);
	bool planeTrace = traceRay(scene.planes, scene.planeCount, origin, dir, ReflectRay, planeHit);
	bool boxTrace = traceRay(scene.boxes, scene.boxCount, origin, dir, ReflectRay, boxHit);
	closestHit = (sphereHit.t < planeHit.t) ? ((sphereHit.t < boxHit.t) ? sphereHit : boxHit) : ((planeHit.t < boxHit.t) ? planeHit : boxHit);


	if (sphereTrace || planeTrace || boxTrace) {
		vec3 col = lighting(closestHit.mat, scene.lights[0].position, scene.lights[0].colour, closestHit.hitPoint, scene.cam.getPosition(), normalise(closestHit.normal), config);

		if (config.reflections && closestHit.objectType == Reflect && depth <= config.maxDepth) {
			vec3 r = normalise(reflect(dir, closestHit.normal));
			vec3 reflectionCol = reflectionCast2(closestHit.hitPoint + closestHit.normal * config.shadowBias, r, scene, config, closestHit.objectPos, depth++);
			if (closestHit.objectName == ObjectName::Plane_t) {
				col += config.planeReflectionStrength * reflectionCol;
			}
			else if (closestHit.objectName == ObjectName::Sphere_t) {
				col += config.sphereReflectionStrength * reflectionCol;
			}
		}

		if (config.renderHardShadows) {
			Hit shadowHit;
			bool shadowTrace = traceRay(scene.spheres, scene.sphereCount, closestHit.hitPoint + closestHit.normal * config.shadowBias, normalise(scene.lights[0].position - closestHit.hitPoint), ShadowRay, shadowHit);
			col = shadowTrace ? col * config.shadowIntensity : col;
		}

		return col;
	}

	return config.backgroundCol;
}

__device__ vec3 rayCast(vec3& origin, vec3& dir, Scene& scene, SceneConfig& config, hiprandState randState) {
	Hit closestHit;
	bool sphereTrace = traceRay(scene.spheres, scene.sphereCount, origin, dir, PrimaryRay, closestHit);
	bool planeTrace = traceRay(scene.planes, scene.planeCount, origin, dir, PrimaryRay, closestHit);
	bool boxTrace = traceRay(scene.boxes, scene.boxCount, origin, dir, PrimaryRay, closestHit);
	bool triangleTrace = traceRay(scene.triangles, scene.triangleCount, origin, dir, PrimaryRay, closestHit);
	bool modelTrace = traceRay(scene.models, scene.modelCount, origin, dir, PrimaryRay, closestHit);

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (sphereTrace || planeTrace || boxTrace || triangleTrace || modelTrace) {
		vec3 col = lighting(closestHit.mat, scene.lights[0].position, scene.lights[0].colour, closestHit.hitPoint, scene.cam.getPosition(), normalise(closestHit.normal), config);

		if (!closestHit.debug) {
			if (config.reflections && closestHit.objectType == Reflect) {
				vec3 r = normalise(reflect(dir, closestHit.normal));
				vec3 reflectionCol = reflectionCast(closestHit.hitPoint + closestHit.normal * config.shadowBias, r, scene, config, closestHit.objectPos);
				
				if (closestHit.objectName == ObjectName::Plane_t) {
					col += config.planeReflectionStrength * reflectionCol;
				}
				else if (closestHit.objectName == ObjectName::Sphere_t) {
					col += config.sphereReflectionStrength * reflectionCol;
				}
				else if (closestHit.objectName == ObjectName::Box_t) {
					col += config.boxReflectionStrength * reflectionCol;
				}
			}

			if (config.renderHardShadows) {
				Hit shadowHit;
				bool sphereShadowTrace = traceRay(scene.spheres, scene.sphereCount, closestHit.hitPoint + closestHit.normal * config.shadowBias, normalise(scene.lights[0].position - closestHit.hitPoint), ShadowRay, shadowHit);
				bool boxShadowTrace = traceRay(scene.boxes, scene.boxCount, closestHit.hitPoint + closestHit.normal * config.shadowBias, normalise(scene.lights[0].position - closestHit.hitPoint), ShadowRay, shadowHit);
				col = (sphereShadowTrace || boxShadowTrace) ? col * config.shadowIntensity : col;
			}

			if (config.renderSoftShadows) {
				int hits = 0;
				for (int i = 0; i < config.softShadowNum; i++) {
					
					// generate random points on a unit sphere
					unsigned int seed = (x * 1000 + y) * 1973 + i * 9277;
					float r = config.softShadowRadius;
					float theta = 2 * 3.141592654f * wangHash(seed);
					float phi = acos(2 * wangHash(seed * 16807) - 1);
					vec3 pointOnSphere = vec3(r * sin(phi) * cos(theta), r * sin(phi) * sin(theta), r * cos(phi));

					// add random points to the light position and test for collision
					vec3 lightPoint = scene.lights[0].position + pointOnSphere;
					Hit hit;
					hits += traceRay(scene.spheres, scene.sphereCount, closestHit.hitPoint + closestHit.normal * config.shadowBias, normalise(lightPoint - closestHit.hitPoint), ShadowRay, hit) ? 1 : 0;
					hits += traceRay(scene.boxes, scene.boxCount, closestHit.hitPoint + closestHit.normal * config.shadowBias, normalise(lightPoint - closestHit.hitPoint), ShadowRay, hit) ? 1 : 0;

				}
				col = col * (1 - ((float)hits / config.softShadowNum));
			}
		}

		return col;
	}

	return vec3(0.1, 0.1, 0.1) * (float)config.backgroundBrightness;
}

__global__ void rayTrace(int width, int height, GLubyte* framebuffer, Scene scene, SceneConfig config, hiprandState* randStates) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int pixelIndex = y * width + x;
	if (x >= width || y >= height) return;

	int id = threadIdx.x + blockIdx.x * blockDim.x;
	hiprandState state = randStates[id];

	vec3 col;

	if (config.antiAliasing) {
		vec3 total;
		vec3 cameraSpacePoint;
		for (float i = 0.25; i <= 0.75; i += 0.5) {
			for (float j = 0.25; j <= 0.75; j += 0.5) {
				cameraSpacePoint = scene.cam.rasterToCameraSpace(float(x + i), float(y + j), width, height);
				total += rayCast(scene.cam.getPosition(), normalise(cameraSpacePoint), scene, config, state);
			}
		}
		col = total / 4;
	}
	else {
		vec3 cameraSpacePoint = scene.cam.rasterToCameraSpace(float(x + 0.5), float(y + 0.5), width, height);
		col = rayCast(scene.cam.getPosition(), normalise(cameraSpacePoint), scene, config, state);
	}

	framebuffer[pixelIndex * 3 + 0] = min(col.x(), 1.0f) * 255;
	framebuffer[pixelIndex * 3 + 1] = min(col.y(), 1.0f) * 255;
	framebuffer[pixelIndex * 3 + 2] = min(col.z(), 1.0f) * 255;
}

__global__ void setupCurand(hiprandState* randStates) {
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	hiprand_init(1337, id, 0, &randStates[id]);
}






//vec3 origin = scene.cam.getPosition();
//vec3 dir = normalise(scene.cam.rasterToCameraSpace(float(x + 0.5), float(y + 0.5), width, height));
//
//vec3 v0 = vec3(0.0, 0.0, 0.0);
//vec3 v1 = vec3(50.0, 0.0, 0.0);
//vec3 v2 = vec3(50.0, 0.0, -50.0);
//
//vec3 AB = v1 - v0;
//vec3 AC = v2 - v0;
//vec3 N = cross(AB, AC);
//
//float rayNormalAngle = dot(N, dir);
//if (abs(rayNormalAngle) < 0.001) {
//	col = config.backgroundCol;
//}
//else {
//	float d = -dot(N, v0);
//	float t = -(dot(N, origin) + d) / rayNormalAngle;
//	if (t < 0) {
//		col = config.backgroundCol;
//	}
//	else {
//		vec3 p = origin + dir * t;
//		vec3 Ne;
//		vec3 v0p = p - v0;
//		Ne = cross(AB, v0p);
//		if (dot(N, Ne) < 0) {
//			col = config.backgroundCol;
//		}
//		else {
//			vec3 CB = v2 - v1;
//			vec3 v1p = p - v1;
//			Ne = cross(CB, v1p);
//			if (dot(N, Ne) < 0) {
//				col = config.backgroundCol;
//			}
//			else {
//				vec3 CA = v0 - v2;
//				vec3 v2p = p - v2;
//				Ne = cross(CA, v2p);
//				if (dot(N, Ne) < 0) {
//					col = config.backgroundCol;
//				}
//				else {
//					col = vec3(1.0, 0.0, 0.0);
//				}
//			}
//		}
//
//	}
//}