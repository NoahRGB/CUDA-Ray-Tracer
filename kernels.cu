#include "hip/hip_runtime.h"
#include "kernels.h"
#include "Object.h"
#include "Camera.h"

#include <stdio.h>

#include <>

__device__ vec3 lighting(CUDAMaterial mat, vec3 lightPos, vec3 lightIntensity, vec3 point, vec3 eye, vec3 normal, SceneConfig& config) {
	vec3 colour;
	vec3 L = normalise(lightPos - point);
	
	// ambient
	if (config.ambientLighting) {
		colour = lightIntensity * mat.colour * mat.ambient;
	}

	// diffuse
	if (config.diffuseLighting) {
		float NdotL = max(dot(normal, L), 0.0f);
		colour += (NdotL > 0.0) ? (mat.colour * lightIntensity * mat.diffuse * NdotL) : vec3();
	}

	// specular
	if (config.specularLighting) {
		vec3 v = normalise(eye);
		vec3 r = normalise(-reflect(L, normal));
		float RdotV = max(dot(r, v), 0.0f);
		colour += mat.specular * lightIntensity * pow(RdotV, mat.shininess);
	}

	return colour;
}

template<typename T> __device__ Hit rayCast(T* objects, int objectCount, vec3 origin, vec3 dir) {
	Hit closestHit;
	
	for (int i = 0; i < objectCount; i++) {
		//skip object
		float t0, t1;
		if (objects[i].hit(origin, dir, t0, t1)) {
			if (t0 < closestHit.t) {
				vec3 hitPoint = origin + t0 * dir;
				vec3 normal = objects[i].normalAt(hitPoint);
				closestHit = { t0, objects[i].mat, hitPoint, normal, objects[i].position };
			}

		}
	}

	closestHit.normal = normalise(closestHit.normal);
	return closestHit;
}

template<typename T> __device__ bool hardShadow(Hit hit, CUDALight* lights, T* objects, int objectCount) {
	for (int i = 0; i < 1; i++) { // for every light
		vec3 dir = normalise(lights[i].position - hit.hitPoint);
		Hit shadowHit = rayCast(objects, objectCount, hit.hitPoint + dir, dir);
		if (shadowHit.hitPoint != vec3(0, 0, 0) && shadowHit.objectPos != hit.objectPos) {
			//Hit test = rayCast(objects, objectCount, hit.hitPoint + dir, dir);
			return true;

		}
	}

	//for (int i = 0; i < 1; i++) { // for every light
	//	vec3 dir = normalise(lights[i].position - hit.hitPoint);
	//	vec3 origin = hit.hitPoint + hit.normal * 9;
	//	Hit shadowHit;

	//	for (int i = 0; i < objectCount; i++) {
	//		//skip object
	//		float t0, t1;
	//		CUDASphere ob = objects[i];


	//		if (objects[i].hit(origin, dir, t0, t1)) {
	//			if (!(t0 < 0 && t1 < 0)) {
	//				float smallest;
	//				if (t0 < 0) {
	//					smallest = t1;
	//				}
	//				else if (t1 < 0) {
	//					smallest = t0;
	//				}
	//				else {
	//					smallest = min(t0, t1);
	//				}

	//				if (smallest < shadowHit.t) {
	//					vec3 hitPoint = origin + smallest * dir;
	//					vec3 normal = objects[i].normalAt(hitPoint);
	//					normal = normalise(normal);//save doing for end

	//					shadowHit = { smallest, objects[i].mat, hitPoint, normal, objects[i].center };
	//				}
	//			}
	//		}
	//	}

	//	if (shadowHit.hitPoint != vec3(0, 0, 0) && shadowHit.objectPos != hit.objectPos) {
	//		//Hit test = rayCast(objects, objectCount, hit.hitPoint + dir, dir);
	//		return true;

	//	}
	//}

	//for (int i = 0; i < 1; i++) {

	//	vec3 origin = hit.hitPoint + hit.normal * 0.5;
	//	vec3 dir = normalise(lights[i].position - hit.hitPoint);
	//	bool hitSomething = false;
	//	for (int j = 0; j < objectCount; j++) {


	//		float t0, t1;
	//		if (objects[j].hit(origin, dir, t0, t1)) {
	//			if (t0 > 0 || t1 > 0) {
	//				if (objects[j].position == hit.objectPos) {
	//					return false;
	//				}
	//				else {
	//					hitSomething = true;
	//				}
	//			}
	//		}
	//	}
	//	if (hitSomething) {
	//		return true;
	//	}
	//}

	return false;
}

__global__ void rayTrace(int width, int height, GLubyte* framebuffer, Scene scene, SceneConfig config) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int pixelIndex = y * width + x;
	if (x >= width || y >= height) return;

	vec3 cameraSpacePoint = scene.cam.rasterToCameraSpace(float(x + 0.5), float(y + 0.5), width, height);
	Hit closestSphereHit = rayCast(scene.spheres, scene.sphereCount, scene.cam.getPosition(), normalise(cameraSpacePoint));
	Hit closestPlaneHit = rayCast(scene.planes, scene.planeCount, scene.cam.getPosition(), normalise(cameraSpacePoint));
	Hit closestHit = (closestSphereHit.t < closestPlaneHit.t) ? closestSphereHit : closestPlaneHit;

	if (closestHit.hitPoint != vec3(0, 0, 0)) {

		vec3 col = lighting(closestHit.mat, scene.lights[0].position, scene.lights[0].colour, closestHit.hitPoint, scene.cam.getPosition(), normalise(closestHit.normal), config);

		if (config.renderHardShadows) {
			if (hardShadow(closestHit, scene.lights, scene.spheres, scene.sphereCount)) {
				col = vec3(0.0, 0.0, 0.0);
			}
		}

		// assign colour value so that it is >= 0 and <= 255
		framebuffer[pixelIndex * 3 + 0] = min(col.x(), 1.0f) * 255;
		framebuffer[pixelIndex * 3 + 1] = min(col.y(), 1.0f) * 255;
		framebuffer[pixelIndex * 3 + 2] = min(col.z(), 1.0f) * 255;
	}
	else {
		// must be a background pixel
		framebuffer[pixelIndex * 3 + 0] = 100.0;
		framebuffer[pixelIndex * 3 + 1] = 100.0;
		framebuffer[pixelIndex * 3 + 2] = 100.0;
	}
}