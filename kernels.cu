#include "hip/hip_runtime.h"
#include "kernels.h"
#include "Object.h"
#include "Camera.h"

#include <stdio.h>
#include <math.h>

#include <>
#include <hiprand/hiprand_kernel.h>

__device__ float wangHash(unsigned int seed) {
	seed = (seed ^ 61) ^ (seed >> 16);
	seed *= 9;
	seed = seed ^ (seed >> 4);
	seed *= 0x27d4eb2d;
	seed = seed ^ (seed >> 15);
	return float(seed) / 4294967296.0f;
}

__device__ vec3 lighting(Material mat, vec3 lightPos, vec3 lightIntensity, vec3 point, vec3 eye, vec3 normal, SceneConfig& config) {
	vec3 colour;
	vec3 L = normalise(lightPos - point);
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	
	// ambient
	if (config.ambientLighting) {
		colour = lightIntensity * mat.ambientColour * mat.ambient;
	}

	// diffuse
	if (config.diffuseLighting) {
		float NdotL = max(dot(normal, L), 0.0f);
		colour += (NdotL > 0.0) ? (mat.diffuseColour * lightIntensity * mat.diffuse * NdotL) : vec3();
	}

	// specular
	if (config.specularLighting) {
		if (config.areaLightSpecularEffect) {
			for (int i = 0; i < config.softShadowNum; i++) {

				unsigned int seed = (x * 1000 + y) * 1973 + i * 9277;
				float radius = config.softShadowRadius;
				float theta = 2 * 3.141592654f * wangHash(seed);
				float phi = acos(2 * wangHash(seed * 16807) - 1);
				vec3 pointOnSphere = vec3(radius * sin(phi) * cos(theta), radius * sin(phi) * sin(theta), radius * cos(phi));
				vec3 L = normalise((lightPos + pointOnSphere) - point);

				vec3 v = normalise(eye);
				vec3 r = normalise(-reflect(L, normal));
				float RdotV = max(dot(r, v), 0.0f);
				colour += mat.specular * lightIntensity * pow(RdotV, mat.shininess);

			}
		}
		else {
			vec3 v = normalise(eye);
			vec3 r = normalise(-reflect(L, normal));
			float RdotV = max(dot(r, v), 0.0f);
			colour += mat.specular * lightIntensity * pow(RdotV, mat.shininess);
		}
	}

	return colour;
}

template<typename T> __device__ bool traceRay(T* objects, int objectCount, vec3 origin, vec3 dir, RayType rayType, Hit& hit, vec3& ignore = vec3(-999, -999, -999)) {
	for (int i = 0; i < objectCount; i++) {

		if (objects[i].position != ignore) {
			float t0, t1;
			if (objects[i].hit(origin, dir, t0, t1)) {

				if (rayType == ShadowRay) {
					if (!objects[i].debug) return true;
					continue;
				}

				if (rayType == ReflectRay && objects[i].debug) continue;

				if (t0 < hit.t) {
					vec3 hitPoint = origin + t0 * dir;
					vec3 normal = objects[i].normalAt(hitPoint);
					hit = { t0, objects[i].mat, hitPoint, normal, objects[i].position, objects[i].objectType, objects[i].objectName, objects[i].debug };
				}
			}
		}
	}

	if (hit.t != 999) {
		hit.normal = normalise(hit.normal);
		return true;
	}

	return false;
}

__device__ bool modelTraceRay(Model* models, int modelCount, vec3 origin, vec3 dir, RayType rayType, Hit& hit, bool accelerate, bool accelerateTwice, bool octree, bool cull) {
	for (int i = 0; i < modelCount; i++) {

		float t0, t1;
		Vertex hitVertex;
		if (models[i].hit(origin, dir, t0, t1, hitVertex, rayType, accelerate, accelerateTwice, octree, cull)) {

			if (rayType == ShadowRay) {
				if (!models[i].debug) return true;
				continue;
			}

			if (rayType == ReflectRay && models[i].debug) continue;

			if (t0 < hit.t) {
				vec3 hitPoint = origin + t0 * dir;
				Material mat = models[i].mat;
				mat.ambientColour = hitVertex.ambient;
				mat.diffuseColour = hitVertex.diffuse;
				hit = { t0, mat, hitPoint, hitVertex.normal, models[i].position, models[i].objectType, models[i].objectName, models[i].debug };
			}
		}
	}

	if (hit.t != 999) {
		hit.normal = normalise(hit.normal);
		return true;
	}

	return false;
}

__device__ vec3 reflectionCast3(vec3& origin, vec3& dir, Scene& scene, SceneConfig& config, vec3& ignore, int depth = 1) {
	Hit closestHit;
	bool sphereTrace = traceRay(scene.spheres, scene.sphereCount, origin, dir, ReflectRay, closestHit);
	bool planeTrace = config.reflectPlanes ? traceRay(scene.planes, scene.planeCount, origin, dir, ReflectRay, closestHit) : false;
	bool AABBTrace = config.renderAABBs ? traceRay(scene.AABBs, scene.AABBCount, origin, dir, ReflectRay, closestHit) : false;
	bool modelTrace = config.renderModels ? modelTraceRay(scene.models, scene.modelCount, origin, dir, ReflectRay, closestHit, config.boundingBox, config.eightBoundingBoxes, config.octree, config.cullBackTriangles) : false;

	if (sphereTrace || planeTrace || AABBTrace || modelTrace) {
		vec3 col = lighting(closestHit.mat, scene.lights[0].position, scene.lights[0].colour, closestHit.hitPoint, scene.cam.getPosition(), normalise(closestHit.normal), config);

		if (config.renderHardShadows) {
			Hit shadowHit;
			bool shadowTrace = traceRay(scene.spheres, scene.sphereCount, closestHit.hitPoint + closestHit.normal * config.shadowBias, normalise(scene.lights[0].position - closestHit.hitPoint), ShadowRay, shadowHit);
			col = shadowTrace ? col * config.shadowIntensity : col;
		}

		return col;
	}

	return config.backgroundCol;
}

__device__ vec3 reflectionCast2(vec3& origin, vec3& dir, Scene& scene, SceneConfig& config, vec3& ignore, int depth = 1) {
	Hit closestHit;
	bool sphereTrace = traceRay(scene.spheres, scene.sphereCount, origin, dir, ReflectRay, closestHit);
	bool planeTrace = config.reflectPlanes ? traceRay(scene.planes, scene.planeCount, origin, dir, ReflectRay, closestHit) : false;
	bool AABBTrace = config.renderAABBs ? traceRay(scene.AABBs, scene.AABBCount, origin, dir, ReflectRay, closestHit) : false;
	bool modelTrace = config.renderModels ? modelTraceRay(scene.models, scene.modelCount, origin, dir, PrimaryRay, closestHit, config.boundingBox, config.eightBoundingBoxes, config.octree, config.cullBackTriangles) : false;

	if (sphereTrace || planeTrace || AABBTrace || modelTrace) {
		vec3 col = lighting(closestHit.mat, scene.lights[0].position, scene.lights[0].colour, closestHit.hitPoint, scene.cam.getPosition(), normalise(closestHit.normal), config);

		if (config.renderHardShadows) {
			Hit shadowHit;
			bool shadowTrace = traceRay(scene.spheres, scene.sphereCount, closestHit.hitPoint + closestHit.normal * config.shadowBias, normalise(scene.lights[0].position - closestHit.hitPoint), ShadowRay, shadowHit);
			col = shadowTrace ? col * config.shadowIntensity : col;
		}

		if (config.reflections && closestHit.objectType == Reflect && depth < config.maxDepth) {
			vec3 r = normalise(reflect(dir, closestHit.normal));
			vec3 reflectionCol = reflectionCast3(closestHit.hitPoint + closestHit.normal * config.shadowBias, r, scene, config, closestHit.objectPos, depth++);
			if (closestHit.objectName == ObjectName::Plane_t) {
				col += config.planeReflectionStrength * reflectionCol;
			}
			else if (closestHit.objectName == ObjectName::Sphere_t) {
				col += config.sphereReflectionStrength * reflectionCol;
			}
			else if (closestHit.objectName == ObjectName::Model_t) {
				col += config.modelReflectionStrength * reflectionCol;
			}
		}

		return col;
	}

	return config.backgroundCol;
}

__device__ vec3 reflectionCast(vec3& origin, vec3& dir, Scene& scene, SceneConfig& config, vec3& ignore, int depth = 1) {
	Hit closestHit;
	bool sphereTrace = traceRay(scene.spheres, scene.sphereCount, origin, dir, ReflectRay, closestHit);
	bool planeTrace = config.reflectPlanes ? traceRay(scene.planes, scene.planeCount, origin, dir, ReflectRay, closestHit) : false;
	bool AABBTrace = config.renderAABBs ? traceRay(scene.AABBs, scene.AABBCount, origin, dir, ReflectRay, closestHit) : false;
	bool modelTrace = config.renderModels ? modelTraceRay(scene.models, scene.modelCount, origin, dir, ReflectRay, closestHit, config.boundingBox, config.eightBoundingBoxes, config.octree, config.cullBackTriangles) : false;

	if (sphereTrace || planeTrace || AABBTrace || modelTrace) {
		vec3 col = lighting(closestHit.mat, scene.lights[0].position, scene.lights[0].colour, closestHit.hitPoint, scene.cam.getPosition(), normalise(closestHit.normal), config);

		if (config.reflections && closestHit.objectType == Reflect && depth < config.maxDepth) {
			vec3 r = normalise(reflect(dir, closestHit.normal));
			vec3 reflectionCol = reflectionCast2(closestHit.hitPoint + closestHit.normal * config.shadowBias, r, scene, config, closestHit.objectPos, depth++);
			if (closestHit.objectName == ObjectName::Plane_t) {
				col += config.planeReflectionStrength * reflectionCol;
			}
			else if (closestHit.objectName == ObjectName::Sphere_t) {
				col += config.sphereReflectionStrength * reflectionCol;
			}
			else if (closestHit.objectName == ObjectName::Model_t) {
				col += config.modelReflectionStrength * reflectionCol;
			}
		}

		if (config.renderHardShadows) {
			Hit shadowHit;
			bool shadowTrace = traceRay(scene.spheres, scene.sphereCount, closestHit.hitPoint + closestHit.normal * config.shadowBias, normalise(scene.lights[0].position - closestHit.hitPoint), ShadowRay, shadowHit);
			col = shadowTrace ? col * config.shadowIntensity : col;
		}

		return col;
	}

	return config.backgroundCol;
}

__device__ vec3 rayCast(vec3& origin, vec3& dir, Scene& scene, SceneConfig& config, hiprandState randState) {
	Hit closestHit;
	bool sphereTrace = traceRay(scene.spheres, scene.sphereCount, origin, dir, PrimaryRay, closestHit);
	bool planeTrace = traceRay(scene.planes, scene.planeCount, origin, dir, PrimaryRay, closestHit);
	bool AABBTrace = config.renderAABBs ? traceRay(scene.AABBs, scene.AABBCount, origin, dir, ReflectRay, closestHit) : false;
	bool modelTrace = config.renderModels ? modelTraceRay(scene.models, scene.modelCount, origin, dir, PrimaryRay, closestHit, config.boundingBox, config.eightBoundingBoxes, config.octree, config.cullBackTriangles) : false;

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (sphereTrace || planeTrace || AABBTrace || modelTrace) {
		vec3 col;

		if (!closestHit.debug) {
			col = vec3(1.0, 0.0, 0.0);

			if (closestHit.objectName == ObjectName::AABB_t) {
				col = lighting(closestHit.mat, scene.lights[0].position, scene.lights[0].colour, closestHit.hitPoint, scene.cam.getPosition(), closestHit.normal, config);
			}
			else {
				col = lighting(closestHit.mat, scene.lights[0].position, scene.lights[0].colour, closestHit.hitPoint, scene.cam.getPosition(), normalise(closestHit.normal), config);
			}
			
			
			if (config.reflections && closestHit.objectType == Reflect) {
				vec3 r = normalise(reflect(dir, closestHit.normal));
				if (!(closestHit.objectName == ObjectName::Plane_t && !config.reflectPlanes)) {
					vec3 reflectionCol = reflectionCast(closestHit.hitPoint + closestHit.normal * config.shadowBias, r, scene, config, closestHit.objectPos);

					if (closestHit.objectName == ObjectName::Plane_t) {
						col += config.planeReflectionStrength * reflectionCol;
					}
					else if (closestHit.objectName == ObjectName::Sphere_t) {
						col += config.sphereReflectionStrength * reflectionCol;
					}
					else if (closestHit.objectName == ObjectName::AABB_t) {
						col += config.AABBReflectionStrength * reflectionCol;
					}
					else if (closestHit.objectName == ObjectName::Model_t) {
						col += config.modelReflectionStrength * reflectionCol;
					}
				}
			}

			if (config.renderHardShadows) {
				Hit shadowHit;
				bool sphereShadowTrace = traceRay(scene.spheres, scene.sphereCount, closestHit.hitPoint + closestHit.normal * config.shadowBias, normalise(scene.lights[0].position - closestHit.hitPoint), ShadowRay, shadowHit);
				bool AABBShadowTrace = traceRay(scene.AABBs, scene.AABBCount, closestHit.hitPoint + closestHit.normal * config.shadowBias, normalise(scene.lights[0].position - closestHit.hitPoint), ShadowRay, shadowHit);
				bool modelShadowTrace = modelTraceRay(scene.models, scene.modelCount, closestHit.hitPoint + closestHit.normal * config.shadowBias, normalise(scene.lights[0].position - closestHit.hitPoint), ShadowRay, shadowHit, config.boundingBox, config.eightBoundingBoxes, config.octree, config.cullBackTriangles);
				if (shadowHit.t >= 0) {
					col = (sphereShadowTrace || AABBShadowTrace || modelShadowTrace) ? col * config.shadowIntensity : col;
				}
				
			}

			if (config.renderSoftShadows) {
				int hits = 0;
				for (int i = 0; i < config.softShadowNum; i++) {
					
					// generate random points on a unit sphere
					unsigned int seed = (x * 1000 + y) * 1973 + i * 9277;
					float r = config.softShadowRadius;

					float num1 = wangHash(seed);
					float num2 = wangHash(seed * 16807);
					//float num1 = (hiprand(&randState) / (float)(0x0FFFFFFFFUL));
					//float num2 = (hiprand(&randState) / (float)(0x0FFFFFFFFUL));
					//float num1 = hiprand_uniform(&randState);
					//float num2 = hiprand_uniform(&randState);
					//printf("%f, %f\n", num1, num2);

					float theta = 2 * 3.141592654f * num1;
					float phi = acos(2 * num2 - 1);
					vec3 pointOnSphere = vec3(r * sin(phi) * cos(theta), r * sin(phi) * sin(theta), r * cos(phi));

					// add random points to the light position and test for collision
					vec3 lightPoint = scene.lights[0].position + pointOnSphere;
					Hit hit;
					hits += traceRay(scene.spheres, scene.sphereCount, closestHit.hitPoint + closestHit.normal * config.shadowBias, normalise(lightPoint - closestHit.hitPoint), ShadowRay, hit) ? 1 : 0;
					hits += traceRay(scene.AABBs, scene.AABBCount, closestHit.hitPoint + closestHit.normal * config.shadowBias, normalise(lightPoint - closestHit.hitPoint), ShadowRay, hit) ? 1 : 0;
					hits += modelTraceRay(scene.models, scene.modelCount, closestHit.hitPoint + closestHit.normal * config.shadowBias, normalise(lightPoint - closestHit.hitPoint), ShadowRay, hit, config.boundingBox, config.eightBoundingBoxes, config.octree, config.cullBackTriangles) ? 1 : 0;
				}
				col = col * (1 - ((float)hits / config.softShadowNum));
			}
		}
		else {
			// is a debug object so just colour it fully
			col = vec3(closestHit.mat.ambientColour.x(), closestHit.mat.ambientColour.y(), closestHit.mat.ambientColour.z());
		}

		return col;
	}

	return config.backgroundCol;
}

__global__ void rayTrace(int width, int height, GLubyte* framebuffer, Scene scene, SceneConfig config, hiprandState* randStates) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int pixelIndex = y * width + x;
	if (x >= width || y >= height) return;

	int id = threadIdx.x + blockIdx.x * blockDim.x;
	hiprandState state = randStates[id];

	vec3 col;

	if (config.antiAliasing) {
		// if anti aliasing is enabled, sample 4 points per pixel and average the colour 
		vec3 total;
		vec3 cameraSpacePoint;
		for (float i = 0.25; i <= 0.75; i += 0.5) {
			for (float j = 0.25; j <= 0.75; j += 0.5) {
				cameraSpacePoint = scene.cam.rasterToCameraSpace(float(x + i), float(y + j), width, height);
				total += rayCast(scene.cam.getPosition(), normalise(cameraSpacePoint), scene, config, state);
			}
		}
		col = total / 4;

		/*vec3 total;
		vec3 cameraSpacePoint;
		for (float i = 0.33; i <= 1.0; i += 0.33) {
			for (float j = 0.33; j <= 1.0; j += 0.33) {
				cameraSpacePoint = scene.cam.rasterToCameraSpace(float(x + j), float(y + i), width, height);
				total += rayCast(scene.cam.getPosition(), normalise(cameraSpacePoint), scene, config, state);
			}
		}
		col = total / 9;*/
	}
	else {
		// if anti aliasing is disabled, just use one sample in the middle of the pixel
		vec3 cameraSpacePoint = scene.cam.rasterToCameraSpace(float(x + 0.5), float(y + 0.5), width, height);
		col = rayCast(scene.cam.getPosition(), normalise(cameraSpacePoint), scene, config, state);
	}

	randStates[id] = state;

	framebuffer[pixelIndex * 3 + 0] = min(col.x(), 1.0f) * 255;
	framebuffer[pixelIndex * 3 + 1] = min(col.y(), 1.0f) * 255;
	framebuffer[pixelIndex * 3 + 2] = min(col.z(), 1.0f) * 255;
}

__global__ void setupCurand(hiprandState* randStates, unsigned long seed) {
	int id = threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * blockDim.y * gridDim.y;
	hiprand_init(seed+id, id, 0, &randStates[id]);
}