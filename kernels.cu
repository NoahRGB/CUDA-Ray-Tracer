#include "hip/hip_runtime.h"
#include "kernels.h"
#include "Camera.h"

#include <stdio.h>

#include <>

__device__ vec3 lighting(CUDAMaterial mat, vec3 lightPos, vec3 lightIntensity, vec3 point, vec3 eye, vec3 normal) {

	// ambient
	vec3 colour = lightIntensity * mat.colour * mat.ambient;

	// diffuse
	vec3 L = normalise(lightPos - point);
	float NdotL = dot(normal, L);
	if (NdotL < 0) NdotL = 0;

	if (NdotL > 0.0) {
		vec3 diffuse = mat.colour * lightIntensity * mat.diffuse * NdotL;
		colour += diffuse;
	}

	//colour = (NdotL > 0.0) ? mat.colour * lightIntensity * mat.diffuse * NdotL : 0;

	// specular
	vec3 v = normalise(eye);
	vec3 r = normalise(-reflect(L, normal));
	float RdotV = dot(r, v);
	if (RdotV < 0) RdotV = 0;

	vec3 specular = mat.specular * lightIntensity * pow(RdotV, mat.shininess);
	colour += specular;

	return colour;
}

__device__ Hit rayCast(CUDASphere* objects, int objectCount, vec3 origin, vec3 dir) {
	Hit closestHit;

	for (int i = 0; i < objectCount; i++) {
		//skip object
		float t0, t1;
		CUDASphere ob = objects[i];
		if (objects[i].hit(origin, dir, t0, t1)) {
			if (!(t0 < 0 && t1 < 0)) {
				float smallest;
				if (t0 < 0) {
					smallest = t1;
				} else if (t1 < 0) {
					smallest = t0;
				}
				else {
					smallest = min(t0, t1);
				}

				if (smallest < closestHit.t) {
					vec3 hitPoint = origin + smallest * dir;
					vec3 normal = objects[i].normalAt(hitPoint);
					normal = normalise(normal);//save doing for end

					closestHit = { smallest, objects[i].mat, hitPoint, normal, objects[i].center };
				}
			}
		}
	}

	return closestHit;
}

__device__ bool hardShadow(Hit hit, CUDALight* lights, CUDASphere* objects, int objectCount) {
	for (int i = 0; i < 1; i++) { // for every light
		vec3 dir = normalise(lights[i].position - hit.hitPoint);
		Hit shadowHit = rayCast(objects, objectCount, hit.hitPoint + dir, dir);
		if (shadowHit.hitPoint != vec3(0, 0, 0) && shadowHit.objectPos != hit.objectPos) {
			//Hit test = rayCast(objects, objectCount, hit.hitPoint + dir, dir);
			return true;

		}

		//vec3 dir = normalise(hit.hitPoint - lights[i].position);
		//Hit shadowHit = rayCast(objects, objectCount, lights[i].position + dir, dir);
		//if (shadowHit.hitPoint != vec3(0, 0, 0) && shadowHit.objectPos != hit.objectPos) {

		//	return true;
		//}
	}
	return false;
}

__global__ void rayTrace(int width, int height, GLubyte* framebuffer, CUDASphere* objects, int objectCount, CUDALight* lights, Camera cam) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int pixelIndex = y * width + x;
	if (x >= width || y >= height) return;

	if (pixelIndex > 250000) {
		int a = 10;
	}


	vec3 cameraSpacePoint = cam.rasterToCameraSpace(float(x + 0.5), float(y + 0.5), width, height);
	Hit closestHit = rayCast(objects, objectCount, cam.getPosition(), normalise(cameraSpacePoint));

	if (closestHit.hitPoint != vec3(0, 0, 0)) {

		vec3 col = lighting(closestHit.mat, lights[0].position, lights[0].colour, closestHit.hitPoint, cam.getPosition(), normalise(closestHit.normal));

		if (hardShadow(closestHit, lights, objects, objectCount)) {
			col = vec3(0.0, 0.0, 0.0);
		}

		//min(col.x, 1.0, col.x);

		if (col.x() > 1) col = vec3(1, col.y(), col.z());
		if (col.y() > 1) col = vec3(col.x(), 1, col.z());
		if (col.z() > 1) col = vec3(col.x(), col.y(), 1);

		framebuffer[pixelIndex * 3 + 0] = 255 * col.x();
		framebuffer[pixelIndex * 3 + 1] = 255 * col.y();
		framebuffer[pixelIndex * 3 + 2] = 255 * col.z();
	}
	else {
		// must be a background pixel
		framebuffer[pixelIndex * 3 + 0] = 100.0;
		framebuffer[pixelIndex * 3 + 1] = 100.0;
		framebuffer[pixelIndex * 3 + 2] = 100.0;
	}
}